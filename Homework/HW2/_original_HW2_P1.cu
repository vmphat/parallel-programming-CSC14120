
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(EXIT_FAILURE);\
    }\
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};


__global__ void reduceBlksKernel1(int * in, int * out, int n)
{
	// TODO
}

__global__ void reduceBlksKernel2(int * in, int * out,int n)
{
	// TODO
}

__global__ void reduceBlksKernel3(int * in, int * out,int n)
{
	// TODO
}

int reduce(int const * in, int n,
        bool useDevice=false, dim3 blockSize=dim3(1), int kernelType=1)
{

	GpuTimer timer;
	int result = 0; // Init
	if (useDevice == false)
	{
		timer.Start();
		result = in[0];
		for (int i = 1; i < n; i++)
		{
			result += in[i];
		}
		timer.Stop();
		float hostTime = timer.Elapsed();
		printf("Host time: %f ms\n",hostTime);
	}
	else // Use device
	{
		// Allocate device memories
		int * d_in, * d_out;
		dim3 gridSize(1); // TODO: Compute gridSize from n and blockSize
		
		// TODO: Allocate device memories

		// TODO: Copy data to device memories

		// Call kernel
		timer.Start();
		if (kernelType == 1)
			reduceBlksKernel1<<<gridSize, blockSize>>>(d_in, d_out, n);
		else if (kernelType == 2)
			reduceBlksKernel2<<<gridSize, blockSize>>>(d_in, d_out, n);
		else 
			reduceBlksKernel3<<<gridSize, blockSize>>>(d_in, d_out, n);

		hipDeviceSynchronize();
		timer.Stop();
		float kernelTime = timer.Elapsed();

		CHECK(hipGetLastError());
		
		// TODO: Copy result from device memories

		// TODO: Free device memories

		// Print info
		printf("\nKernel %d\n", kernelType);
		printf("Grid size: %d, block size: %d\n", gridSize.x, blockSize.x);
		printf("Kernel time = %f ms\n", kernelTime);
	}

	return result;
}

void checkCorrectness(int r1, int r2)
{
	if (r1 == r2)
		printf("CORRECT :)\n");
	else
		printf("INCORRECT :(\n");
}

void printDeviceInfo()
{
	hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
    printf("****************************\n\n");

}

int main(int argc, char ** argv)
{
	printDeviceInfo();

	// Set up input size
    int n = (1 << 24)+1;
    printf("Input size: %d\n", n);

    // Set up input data
    int * in = (int *) malloc(n * sizeof(int));
    for (int i = 0; i < n; i++)
    {
        // Generate a random integer in [0, 255]
        in[i] = (int)(rand() & 0xFF);
    }

    // Reduce NOT using device
    int correctResult = reduce(in, n);

    // Reduce using device, kernel1
    dim3 blockSize(512); // Default
    if (argc == 2)
    	blockSize.x = atoi(argv[1]); 
 	
	int result1 = reduce(in, n, true, blockSize, 1);
    checkCorrectness(result1, correctResult);

    // Reduce using device, kernel2
    int result2 = reduce(in, n, true, blockSize, 2);
    checkCorrectness(result2, correctResult);

    // Reduce using device, kernel3
    int result3 = reduce(in, n, true, blockSize, 3);
    checkCorrectness(result3, correctResult);

    // Free memories
    free(in);
}
