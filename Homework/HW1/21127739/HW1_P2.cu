#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>

#define CHECK(call)\
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", error,\
				hipGetErrorString(error));\
		exit(EXIT_FAILURE);\
	}\
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void readPnm(char * fileName, 
		int &width, int &height, uchar3 * &pixels)
{
	FILE * f = fopen(fileName, "r");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);
	
	if (strcmp(type, "P3") != 0) // In this exercise, we don't touch other types
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	fscanf(f, "%i", &width);
	fscanf(f, "%i", &height);
	
	int max_val;
	fscanf(f, "%i", &max_val);
	if (max_val > 255) // In this exercise, we assume 1 byte per value
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	for (int i = 0; i < width * height; i++)
		fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

	fclose(f);
}

void writePnm(uchar3 * pixels, int width, int height, 
		char * fileName)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	fprintf(f, "P3\n%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height; i++)
		fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);
	
	fclose(f);
}

__global__ void blurImgKernel(uchar3 * inPixels, int width, int height, 
		float * filter, int filterWidth, 
		uchar3 * outPixels)
{
	// TODO
	
    // Calculate the row and column index of the pixel
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

	// Check if the pixel coordinate is within the image
	if (r < height && c < width)
	{
		// Weighted sum for each color channel for current pixel
		float sumRed   = 0.0f;
		float sumGreen = 0.0f;
		float sumBlue  = 0.0f;

		// Loop through each coordinate in the filter
		for (int filterR = 0; filterR < filterWidth; ++filterR)
		{
			for (int filterC = 0; filterC < filterWidth; ++filterC)
			{
				// Find relative image coordinate for current filter coordinate
				int relativeImageR = r - filterWidth / 2 + filterR;
				int relativeImageC = c - filterWidth / 2 + filterC;

				// Clamp relative image coordinate to image border
				int imageR = min(max(relativeImageR, 0), height - 1);
				int imageC = min(max(relativeImageC, 0), width  - 1);

				// Find the valid global image index
				int imageIdx = imageR * width + imageC;

				// Find the global filter index
				int filterIdx = filterR * filterWidth + filterC;

				// Update the weighted sum for each color channel
				sumRed   += inPixels[imageIdx].x * filter[filterIdx];
				sumGreen += inPixels[imageIdx].y * filter[filterIdx];
				sumBlue  += inPixels[imageIdx].z * filter[filterIdx];
			}
		}

		// Update the blurred output pixel with the weighted sum
		// Reference: Struct `uchar3` definition - https://github.com/tpn/cuda-samples/blob/master/v8.0/include/hip/hip_vector_types.h
		int i = r * width + c;
		outPixels[i].x = (unsigned char)sumRed;
		outPixels[i].y = (unsigned char)sumGreen;
		outPixels[i].z = (unsigned char)sumBlue;
	}
}

void blurImg(uchar3 * inPixels, int width, int height, float * filter, int filterWidth, 
		uchar3 * outPixels,
		bool useDevice=false, dim3 blockSize=dim3(1, 1))
{
	GpuTimer timer;
	timer.Start();
	if (useDevice == false)
	{
		// TODO
		
		// Loop through each pixel in the original image
		for (int r = 0; r < height; ++r)
		{
			for (int c = 0; c < width; ++c)
			{
				// Weighted sum for each color channel for current pixel
				float sumRed   = 0.0f;
				float sumGreen = 0.0f;
				float sumBlue  = 0.0f;

				// Loop through each coordinate in the filter
				for (int filterR = 0; filterR < filterWidth; ++filterR)
				{
					for (int filterC = 0; filterC < filterWidth; ++filterC)
					{
						// Find relative image coordinate for current filter coordinate
						int relativeImageR = r - filterWidth / 2 + filterR;
						int relativeImageC = c - filterWidth / 2 + filterC;

						// Clamp relative image coordinate to image border
						int imageR = min(max(relativeImageR, 0), height - 1);
						int imageC = min(max(relativeImageC, 0), width  - 1);

						// Find the valid global image index
						int imageIdx = imageR * width + imageC;

						// Find the global filter index
						int filterIdx = filterR * filterWidth + filterC;

						// Update the weighted sum for each color channel
						sumRed   += inPixels[imageIdx].x * filter[filterIdx];
						sumGreen += inPixels[imageIdx].y * filter[filterIdx];
						sumBlue  += inPixels[imageIdx].z * filter[filterIdx];
					}
				}

				// Update the blurred output pixel with the weighted sum
				// Reference: Struct `uchar3` definition - https://github.com/tpn/cuda-samples/blob/master/v8.0/include/hip/hip_vector_types.h
				int i = r * width + c;
				outPixels[i].x = (unsigned char)sumRed;
				outPixels[i].y = (unsigned char)sumGreen;
				outPixels[i].z = (unsigned char)sumBlue;
			}
		}
	}
	else // Use device
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, 0);
		printf("GPU name: %s\n", devProp.name);
		printf("GPU compute capability: %d.%d\n", devProp.major, devProp.minor);

		// TODO

		// Allocate device memories
		uchar3 *d_inPixels, *d_outPixels;
		float *d_filter;		
		size_t numBytesImage  = width * height * sizeof(uchar3);
		size_t numBytesFilter = filterWidth * filterWidth * sizeof(float);
		CHECK(hipMalloc((void **)&d_inPixels , numBytesImage ));
		CHECK(hipMalloc((void **)&d_outPixels, numBytesImage ));
		CHECK(hipMalloc((void **)&d_filter   , numBytesFilter));

		// Copy data to device memories
		CHECK(hipMemcpy(d_inPixels, inPixels, numBytesImage , hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_filter  , filter  , numBytesFilter, hipMemcpyHostToDevice));

		// Set grid size and call kernel (also check kernel error)
		dim3 gridSize((width  - 1) / blockSize.x + 1, 
					  (height - 1) / blockSize.y + 1);
		blurImgKernel<<<gridSize, blockSize>>>(d_inPixels, width, height, d_filter, filterWidth, d_outPixels);
		// Ensure all device operations are complete
		// Reference: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__DEVICE.html#group__CUDART__DEVICE_1g10e20b05a95f638a4071a655503df25d
		CHECK(hipDeviceSynchronize());
		// Check for kernel launch errors
		// Reference: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__ERROR.html#group__CUDART__ERROR
		CHECK(hipGetLastError());

		// Copy result from device memories
		CHECK(hipMemcpy(outPixels, d_outPixels, numBytesImage, hipMemcpyDeviceToHost));

		// Free device memories
		CHECK(hipFree(d_inPixels));
		CHECK(hipFree(d_outPixels));
		CHECK(hipFree(d_filter));
	}
	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time (%s): %f ms\n", 
    		useDevice == true? "use device" : "use host", time);
}

float computeError(uchar3 * a1, uchar3 * a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)
	{
		err += abs((int)a1[i].x - (int)a2[i].x);
		err += abs((int)a1[i].y - (int)a2[i].y);
		err += abs((int)a1[i].z - (int)a2[i].z);
	}
	err /= (n * 3);
	return err;
}

char * concatStr(const char * s1, const char * s2)
{
    char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

int main(int argc, char ** argv)
{
	if (argc != 4 && argc != 6)
	{
		printf("The number of arguments is invalid\n");
		return EXIT_FAILURE;
	}

	// Read input image file
	int width, height;
	uchar3 * inPixels;
	readPnm(argv[1], width, height, inPixels);
	printf("Image size (width x height): %i x %i\n\n", width, height);

	// Read correct output image file
	int correctWidth, correctHeight;
	uchar3 * correctOutPixels;
	readPnm(argv[3], correctWidth, correctHeight, correctOutPixels);
	if (correctWidth != width || correctHeight != height)
	{
		printf("The shape of the correct output image is invalid\n");
		return EXIT_FAILURE;
	}

	// Set up a simple filter with blurring effect 
	int filterWidth = 9;
	float * filter = (float *)malloc(filterWidth * filterWidth * sizeof(float));
	for (int filterR = 0; filterR < filterWidth; filterR++)
	{
		for (int filterC = 0; filterC < filterWidth; filterC++)
		{
			filter[filterR * filterWidth + filterC] = 1. / (filterWidth * filterWidth);
		}
	}

	// Blur input image using host
	uchar3 * hostOutPixels = (uchar3 *)malloc(width * height * sizeof(uchar3)); 
	blurImg(inPixels, width, height, filter, filterWidth, hostOutPixels);
	
	// Compute mean absolute error between host result and correct result
	float hostErr = computeError(hostOutPixels, correctOutPixels, width * height);
	printf("Error: %f\n\n", hostErr);

	// Blur input image using device
	uchar3 * deviceOutPixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	dim3 blockSize(32, 32); // Default
	if (argc == 6)
	{
		blockSize.x = atoi(argv[4]);
		blockSize.y = atoi(argv[5]);
	}  
	blurImg(inPixels, width, height, filter, filterWidth, deviceOutPixels, true, blockSize);

	// Compute mean absolute error between device result and correct result
	float deviceErr = computeError(deviceOutPixels, correctOutPixels, width * height);
	printf("Error: %f\n\n", deviceErr);

	// Write results to files
	char * outFileNameBase = strtok(argv[2], "."); // Get rid of extension
	writePnm(hostOutPixels, width, height, concatStr(outFileNameBase, "_host.pnm"));
	writePnm(deviceOutPixels, width, height, concatStr(outFileNameBase, "_device.pnm"));

	// Free memories
	free(inPixels);
	free(correctOutPixels);
	free(hostOutPixels);
	free(deviceOutPixels);
	free(filter);
}
