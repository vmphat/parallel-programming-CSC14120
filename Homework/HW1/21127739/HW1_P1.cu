
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define CHECK(call)\
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", error,\
				hipGetErrorString(error));\
		exit(EXIT_FAILURE);\
	}\
}

struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start, 0);                                                                 
		hipEventSynchronize(start);
	}

	void Stop()
	{
		hipEventRecord(stop, 0);
	}

	float Elapsed()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};

void readPnm(char * fileName, 
		int &numChannels, int &width, int &height, uint8_t * &pixels)
{
	FILE * f = fopen(fileName, "r");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);
	if (strcmp(type, "P2") == 0)
		numChannels = 1;
	else if (strcmp(type, "P3") == 0)
		numChannels = 3;
	else // In this exercise, we don't touch other types
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	fscanf(f, "%i", &width);
	fscanf(f, "%i", &height);

	int max_val;
	fscanf(f, "%i", &max_val);
	if (max_val > 255) // In this exercise, we assume 1 byte per value
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	pixels = (uint8_t *)malloc(width * height * numChannels);
	for (int i = 0; i < width * height * numChannels; i++)
		fscanf(f, "%hhu", &pixels[i]);

	fclose(f);
}

void writePnm(uint8_t * pixels, int numChannels, int width, int height, 
		char * fileName)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	if (numChannels == 1)
		fprintf(f, "P2\n");
	else if (numChannels == 3)
		fprintf(f, "P3\n");
	else
	{
		fclose(f);
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	fprintf(f, "%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height * numChannels; i++)
		fprintf(f, "%hhu\n", pixels[i]);

	fclose(f);
}

__global__ void convertRgb2GrayKernel(uint8_t * inPixels, int width, int height, 
		uint8_t * outPixels)
{
	// TODO
    // Reminder: gray = 0.299*red + 0.587*green + 0.114*blue  

    // Calculate the row and column index of the pixel
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

	// If the pixel coordinate is within the image
    if (row < height && col < width)
    {
		// Get 1D offset for the grayscale image
		int grayOffset = row * width + col;

        // Get 1D offset for the RGB image
		// Each pixel in RGB image has 3 values: red, green, and blue
		int rgbOffset = 3 * grayOffset;
		// Get red, green, and blue values of the pixel
        uint8_t red   = inPixels[rgbOffset    ];
        uint8_t green = inPixels[rgbOffset + 1];
        uint8_t blue  = inPixels[rgbOffset + 2];

		// Perform the rescaling and store the result
        outPixels[grayOffset] = 0.299f*red + 0.587f*green + 0.114f*blue;
    }
}

void convertRgb2Gray(uint8_t * inPixels, int width, int height,
		uint8_t * outPixels, 
		bool useDevice=false, dim3 blockSize=dim3(1))
{
	GpuTimer timer;
	timer.Start();
	if (useDevice == false)
	{
        // Reminder: gray = 0.299*red + 0.587*green + 0.114*blue  
        for (int r = 0; r < height; r++)
        {
            for (int c = 0; c < width; c++)
            {
                int i = r * width + c;
                uint8_t red = inPixels[3 * i];
                uint8_t green = inPixels[3 * i + 1];
                uint8_t blue = inPixels[3 * i + 2];
                outPixels[i] = 0.299f*red + 0.587f*green + 0.114f*blue;
            }
        }
	}
	else // use device
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, 0);
		printf("GPU name: %s\n", devProp.name);
		printf("GPU compute capability: %d.%d\n", devProp.major, devProp.minor);

		// TODO: Allocate device memories
		uint8_t *d_inPixels, *d_outPixels;
		// Number of bytes for the RGB image and the grayscale image
		size_t numBytesRgb  = width * height * 3 * sizeof(uint8_t);
		size_t numBytesGray = width * height * sizeof(uint8_t);
        CHECK(hipMalloc((void **)&d_inPixels , numBytesRgb ));
        CHECK(hipMalloc((void **)&d_outPixels, numBytesGray));

		// TODO: Copy data to device memories
        CHECK(hipMemcpy(d_inPixels, inPixels, numBytesRgb, hipMemcpyHostToDevice));

		// TODO: Set grid size and call kernel (remember to check kernel error)
        dim3 gridSize((width  - 1) / blockSize.x + 1, 
                      (height - 1) / blockSize.y + 1);
        convertRgb2GrayKernel<<<gridSize, blockSize>>>(d_inPixels, width, height, d_outPixels);
		// Ensure all device operations are complete
		// Reference: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__DEVICE.html#group__CUDART__DEVICE_1g10e20b05a95f638a4071a655503df25d
		CHECK(hipDeviceSynchronize());
		// Check for kernel launch errors
		// Reference: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__ERROR.html#group__CUDART__ERROR
		CHECK(hipGetLastError());

		// TODO: Copy result from device memories
        CHECK(hipMemcpy(outPixels, d_outPixels, numBytesGray, hipMemcpyDeviceToHost));

		// TODO: Free device memories
        CHECK(hipFree(d_inPixels));
        CHECK(hipFree(d_outPixels));
	}
	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time (%s): %f ms\n\n", 
			useDevice == true? "use device" : "use host", time);
}

float computeError(uint8_t * a1, uint8_t * a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)
		err += abs((int)a1[i] - (int)a2[i]);
	err /= n;
	return err;
}

char * concatStr(const char * s1, const char * s2)
{
	char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
	strcpy(result, s1);
	strcat(result, s2);
	return result;
}

int main(int argc, char ** argv)
{	
	if (argc != 3 && argc != 5)
	{
		printf("The number of arguments is invalid\n");
		return EXIT_FAILURE;
	}

	// Read input RGB image file
	int numChannels, width, height;
	uint8_t * inPixels;
	readPnm(argv[1], numChannels, width, height, inPixels);
	if (numChannels != 3)
		return EXIT_FAILURE; // Input image must be RGB
	printf("Image size (width x height): %i x %i\n\n", width, height);

	// Convert RGB to grayscale not using device
	uint8_t * correctOutPixels= (uint8_t *)malloc(width * height);
	convertRgb2Gray(inPixels, width, height, correctOutPixels);

	// Convert RGB to grayscale using device
	uint8_t * outPixels= (uint8_t *)malloc(width * height);
	dim3 blockSize(32, 32); // Default
	if (argc == 5)
	{
		blockSize.x = atoi(argv[3]);
		blockSize.y = atoi(argv[4]);
	} 
	convertRgb2Gray(inPixels, width, height, outPixels, true, blockSize); 

	// Compute mean absolute error between host result and device result
	float err = computeError(outPixels, correctOutPixels, width * height);
	printf("Error between device result and host result: %f\n", err);

	// Write results to files
	char * outFileNameBase = strtok(argv[2], "."); // Get rid of extension
	writePnm(correctOutPixels, 1, width, height, concatStr(outFileNameBase, "_host.pnm"));
	writePnm(outPixels, 1, width, height, concatStr(outFileNameBase, "_device.pnm"));

	// Free memories
	free(inPixels);
	free(outPixels);
}
