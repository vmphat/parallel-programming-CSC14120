/*‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾*\
|   Họ và tên: Vũ Minh Phát                     |
|   MSSV: 21127739                              |
|   Lớp: Lập trình song song - 21KHMT           |
|   HW0: Làm quen với CUDA                      |
+-----------------------------------------------+
|   Câu 2: Viết chương trình cộng hai vector,   |
|           mỗi thread thực hiện hai phép tính  |
|           cộng trên hai phần tử của mảng.     |
\*_____________________________________________*/

#include <stdio.h>
#include <hip/hip_runtime.h>

// Macro để kiểm tra lỗi sau khi gọi hàm CUDA API.
// Nếu có lỗi, nó sẽ in thông báo lỗi và thoát chương trình.
#define CUDA_CHECK_ERROR(call) {                                \
    const hipError_t err = call;                               \
    if (err != hipSuccess) {                                   \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);  \
        fprintf(stderr, "code: %d, reason: %s\n",               \
                err, hipGetErrorString(err));                  \
        exit(EXIT_FAILURE);                                     \
    }                                                           \
}

// Kích thước block được thống nhất để chạy thử nghiệm
#define BLOCK_SIZE 256
// Số phần tử mỗi thread xử lý
#define ELEMENTS_PER_THREAD 2
// Giá trị dummy để "làm nóng" GPU
#define DUMMY_VALUE 1
// In các dòng kẻ của bảng kết quả
#define OUTER_LINE printf("*-------------*-------------------*-------------------*-------------------*\n");
#define INNER_LINE printf("*-------------+-------------------+-------------------+-------------------*\n");

// Enum để chọn loại thiết bị (host hoặc device)
enum DeviceType {
    HOST,       // Chạy trên host (CPU)
    DEVICE,     // Chạy trên device (GPU)
};
// Enum để chọn version của kernel (chỉ dùng cho device)
enum KernelVersion {
    VERSION_1,  // Gọi hàm kernel `addVecOnDeviceV1`
    VERSION_2,  // Gọi hàm kernel `addVecOnDeviceV2`
    NONE,       // Không chọn version nào
};

/**
 * Cấu trúc để tính thời gian chạy của chương trình
 *
 * Reference: File demo `01-AddVector.cu` được 
 *              cung cấp trong môn học.
 */
struct GpuTimer {
    hipEvent_t start;
    hipEvent_t stop;

    // Khởi tạo cấu trúc GpuTimer
    GpuTimer() {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    // Hủy cấu trúc GpuTimer
    ~GpuTimer() {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    // Bắt đầu tính thời gian
    void Start() {
        hipEventRecord(start, 0);                                                                 
        hipEventSynchronize(start);
    }

    // Kết thúc tính thời gian
    void Stop() {
        hipEventRecord(stop, 0);
    }

    // Trả về thời gian chênh lệch giữa `start` và `stop`
    float Elapsed() {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

/**
 * Hàm cộng hai mảng (vector) trên host (CPU).
 *
 * Ta dùng một vòng lặp để cộng từng phần tử của 
 *  hai mảng và lưu kết quả vào mảng thứ ba.
 *
 * @param in1: con trỏ đến mảng thứ nhất
 * @param in2: con trỏ đến mảng thứ hai
 * @param out: con trỏ đến mảng lưu kết quả
 * @param n: số phần tử của mỗi mảng
 */
void addVecOnHost(const float *in1, const float *in2, float *out, const size_t n) {
    for (size_t i = 0; i < n; ++i) {
        out[i] = in1[i] + in2[i];
    }
}

/**
 * Hàm cộng hai mảng (vector) trên device (GPU) theo version 1.
 * 
 * Mỗi thread block xử lý `ELEMENTS_PER_THREAD * blockDim.x` 
 *  phần tử liên tiếp. Tất cả các thread trong mỗi block sẽ xử lý
 *  `blockDim.x` phần tử đầu mảng, mỗi thread xử lý một phần tử. 
 *  Sau đó tất cả các thread sẽ lần lượt chuyển sang `blockDim.x`
 *  phần tử tiếp theo của mảng, mỗi thread xử lý một phần tử.
 *  Quá trình này lặp lại cho đến khi hết mảng. 
 *
 * @param in1: con trỏ đến mảng thứ nhất trên device
 * @param in2: con trỏ đến mảng thứ hai trên device
 * @param out: con trỏ đến mảng lưu kết quả trên device
 * @param n: số phần tử của mỗi mảng
 */
__global__ 
void addVecOnDeviceV1(const float *in1, const float *in2, float *out, const size_t n) {
    // Số lượng phần tử mỗi (thread-)block có thể xử lý
    size_t elementsPerBlock = ELEMENTS_PER_THREAD * blockDim.x;
    // Index của phần tử đầu tiên mà (thread-)block này xử lý
    size_t blockStartIndex  = blockIdx.x * elementsPerBlock;
    // Index của phần tử đầu tiên mà thread này xử lý
    size_t threadStartIndex = blockStartIndex + threadIdx.x;

    // Mỗi thread xử lý `ELEMENTS_PER_THREAD` phần tử được mô tả như sau:
    //  - Phần tử thứ 1: `threadStartIndex`
    //  - Phần tử thứ 2: `threadStartIndex + blockDim.x`
    //  - Phần tử thứ 3: `threadStartIndex + 2 * blockDim.x`
    //  - ...
    for (size_t i = 0; i < ELEMENTS_PER_THREAD; ++i) {
        size_t index = threadStartIndex + i * blockDim.x;
        if (index >= n) { return; }
        out[index] = in1[index] + in2[index];
    }
}

/**
 * Hàm cộng hai mảng (vector) trên device (GPU) theo version 2.
 *
 * Mỗi thread block xử lý `ELEMENTS_PER_THREAD * blockDim.x` phần tử 
 *  liên tiếp. Mỗi thread sẽ xử lý `ELEMENTS_PER_THREAD` phần tử 
 *  liên tiếp nhau trong mảng.
 *
 * @param in1: con trỏ đến mảng thứ nhất trên device
 * @param in2: con trỏ đến mảng thứ hai trên device
 * @param out: con trỏ đến mảng lưu kết quả trên device
 * @param n: số phần tử của mỗi mảng
 */
__global__ 
void addVecOnDeviceV2(const float *in1, const float *in2, float *out, const size_t n) {
    // Global index của thread này (tương đương với index của phần tử mảng)
    size_t globalThreadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    // Index của phần tử đầu tiên mà thread này xử lý
    size_t threadStartIndex  = globalThreadIndex * ELEMENTS_PER_THREAD;

    // Mỗi thread xử lý `ELEMENTS_PER_THREAD` phần tử được mô tả như sau:
    //  - Phần tử thứ 1: `threadStartIndex`
    //  - Phần tử thứ 2: `threadStartIndex + 1`
    //  - Phần tử thứ 3: `threadStartIndex + 2`
    //  - ...
    for (size_t i = 0; i < ELEMENTS_PER_THREAD; ++i) {
        size_t index = threadStartIndex + i;
        if (index >= n) { return; }
        out[index] = in1[index] + in2[index];
    }
}

float addVec(const float *in1, const float *in2, float *out, const size_t n,
             DeviceType device, KernelVersion version = KernelVersion::NONE) {
    // Khởi tạo timer để đo thời gian chạy
    GpuTimer timer;

    switch(device) {
    // ==================== CHẠY TRÊN HOST ====================
    case DeviceType::HOST: {
        timer.Start();
        addVecOnHost(in1, in2, out, n);
        timer.Stop();
        break;
    }
    // =================== CHẠY TRÊN DEVICE ===================
    case DeviceType::DEVICE: {
        // Host cấp phát bộ nhớ cho các mảng trên device
        float *d_in1, *d_in2, *d_out;
        size_t nBytes = n * sizeof(float);
        CUDA_CHECK_ERROR(hipMalloc((void **)&d_in1, nBytes));
        CUDA_CHECK_ERROR(hipMalloc((void **)&d_in2, nBytes));
        CUDA_CHECK_ERROR(hipMalloc((void **)&d_out, nBytes));
        
        // Copy dữ liệu của mảng đầu vào từ host sang device
        CUDA_CHECK_ERROR(hipMemcpy(d_in1, in1, nBytes, hipMemcpyHostToDevice));
        CUDA_CHECK_ERROR(hipMemcpy(d_in2, in2, nBytes, hipMemcpyHostToDevice));

        // Xác định hàm kernel cần chạy
        void (*kernelFunc)(const float *, const float *, float *, const size_t);
        if (version == KernelVersion::VERSION_1) {
            kernelFunc = addVecOnDeviceV1;
        } 
        else if (version == KernelVersion::VERSION_2) {
            kernelFunc = addVecOnDeviceV2;
        }
        else {
            fprintf(stderr, "Error: Invalid kernel version\n");
            exit(EXIT_FAILURE);
        }

        // Xác định kích thước của grid và block
        size_t elementsPerBlock = ELEMENTS_PER_THREAD * BLOCK_SIZE;
        dim3 blockSize(BLOCK_SIZE);
        dim3 gridSize((n - 1) / elementsPerBlock + 1);

        // Chạy hàm kernel và đo thời gian chạy
        timer.Start();
        kernelFunc<<<gridSize, blockSize>>>(d_in1, d_in2, d_out, n);
        hipDeviceSynchronize(); // Host waits here until device completes its work
        timer.Stop();
        
        // Copy kết quả của mảng đầu ra từ device trở lại host
        CUDA_CHECK_ERROR(hipMemcpy(out, d_out, nBytes, hipMemcpyDeviceToHost));

        // Host giải phóng bộ nhớ cho các mảng trên device sau khi kết thúc
        CUDA_CHECK_ERROR(hipFree(d_in1));
        CUDA_CHECK_ERROR(hipFree(d_in2));
        CUDA_CHECK_ERROR(hipFree(d_out));
        
        break;
    }
    // ============== LOẠI THIẾT BỊ KHÔNG HỢP LỆ ==============
    default: {
        fprintf(stderr, "Error: Invalid device type\n");
        exit(EXIT_FAILURE);
    }
    }

    // Trả về thời gian chạy
    return timer.Elapsed();
}

int main(int argc, char **argv) {
    // Các kích thước mảng (N) khác nhau được dùng để thử nghiệm
    const size_t VECTOR_SIZES[] = {
        DUMMY_VALUE, // Dùng giá trị dummy để "làm nóng" GPU
        64, 256, 1024, 4096, 16384, 
        65536, 262144, 1048576, 4194304, 16777216
    };

    // Số trường hợp cần thử nghiệm
    size_t nTestCases = sizeof(VECTOR_SIZES) / sizeof(VECTOR_SIZES[0]);
    // Khởi tạo các mảng chứa kết quả của từng trường hợp thử nghiệm
    float *hostTimes     = (float *)malloc(nTestCases * sizeof(float));
    float *deviceTimesV1 = (float *)malloc(nTestCases * sizeof(float));
    float *deviceTimesV2 = (float *)malloc(nTestCases * sizeof(float));
    

    // +========================================================+
    // |             In thông tin của card màn hình             |
    // +--------------------------------------------------------+
    hipDeviceProp_t deviceProp;
    CUDA_CHECK_ERROR(hipGetDeviceProperties(&deviceProp, 0));
    printf("===== GPU Information =====\n");
    printf("GPU name: %s\n", deviceProp.name);
    printf("GPU compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
    printf("===========================\n\n");


    // +========================================================+
    // |   Chạy thử nghiệm với các kích thước mảng khác nhau    |
    // +--------------------------------------------------------+
    printf("________[ Running test cases ]________\n");
    for (size_t caseIdx = 0; caseIdx < nTestCases; ++caseIdx) {
        // Kích thước mảng đầu vào
        size_t N = VECTOR_SIZES[caseIdx];
        fprintf(stderr, "Vector size N = %zu ... ", N);

        /**
         * Cấp phát bộ nhớ cho các mảng trên host
         *  - `in1`, `in2`: mảng đầu vào
         *  - `correctOut`: mảng lưu kết quả đúng (tính toán trên host)
         *  - `outV1`, `outV2`: mảng lưu kết quả tính toán trên device
         */
        size_t nBytes     = N * sizeof(float);
        float *in1        = (float *)malloc(nBytes);
        float *in2        = (float *)malloc(nBytes);
        float *correctOut = (float *)malloc(nBytes);
        float *outV1      = (float *)malloc(nBytes);
        float *outV2      = (float *)malloc(nBytes);

        // Khởi tạo mảng đầu vào với giá trị ngẫu nhiên trong đoạn [0, 1]
        for (size_t i = 0; i < N; ++i) {
            in1[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
            in2[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
        }

        // Cộng hai mảng trên host
        float hostTime     = addVec(in1, in2, correctOut, N, DeviceType::HOST);

        // Cộng hai mảng trên device theo version 1
        float deviceTimeV1 = addVec(in1, in2, outV1, N, DeviceType::DEVICE, KernelVersion::VERSION_1);
        
        // Cộng hai mảng trên device theo version 2
        float deviceTimeV2 = addVec(in1, in2, outV2, N, DeviceType::DEVICE, KernelVersion::VERSION_2);

        // Kiểm tra kết quả tính toán trên host và device (cả hai version)
        bool isCorrectV1 = true, isCorrectV2 = true;
        for (size_t i = 0; i < N; ++i) {
            if (correctOut[i] != outV1[i]) {
                isCorrectV1 = false;
                break;
            }
            if (correctOut[i] != outV2[i]) {
                isCorrectV2 = false;
                break;
            }
        }

        // Giải phóng bộ nhớ cho các mảng sau mỗi lần thử nghiệm
        free(in1);
        free(in2);
        free(correctOut);
        free(outV1);
        free(outV2);

        // Kiểm tra tính đúng đắn của kết quả
        if (!isCorrectV1 || !isCorrectV2) {
            fprintf(stderr, "[FAILED] Incorrect result!\n");
            exit(EXIT_FAILURE);
        } 
        else {
            printf("passed\n");
        }

        // Lưu kết quả thời gian chạy vào mảng chứa kết quả
        hostTimes[caseIdx]     = hostTime;
        deviceTimesV1[caseIdx] = deviceTimeV1;
        deviceTimesV2[caseIdx] = deviceTimeV2;
    }
    printf("‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾‾\n\n");


    // +========================================================+
    // |   Hiển thị bảng kết quả tổng hợp của các trường hợp    |
    // +--------------------------------------------------------+
    printf("[===== Result summary =====]\n");
    // In tiêu đề của bảng kết quả tổng hợp
    OUTER_LINE;
    printf("| %-12s| %-18s| %-18s| %-18s|\n", "Vector size", " Host time (ms)", "Device time (ms)", "Device time (ms)");
    printf("| %-12s| %-18s| %-18s| %-18s|\n", " ", " ", "   (Version 1)", "   (Version 2)");
    INNER_LINE;
    // In kết quả (thời gian chạy) của từng trường hợp thử nghiệm
    for (size_t caseIdx = 1; caseIdx < nTestCases; ++caseIdx) {
        printf("| %-12zu| %-18.3f| %-18.3f| %-18.3f|\n",
                VECTOR_SIZES[caseIdx], hostTimes[caseIdx], 
                deviceTimesV1[caseIdx], deviceTimesV2[caseIdx]);
    }
    // In dòng kẻ cuối cùng của bảng kết quả tổng hợp
    OUTER_LINE;

    // Giải phóng bộ nhớ cho các mảng chứa kết quả sau quá trình thử nghiệm
    free(hostTimes);
    free(deviceTimesV1);
    free(deviceTimesV2);

    return EXIT_SUCCESS;
}
