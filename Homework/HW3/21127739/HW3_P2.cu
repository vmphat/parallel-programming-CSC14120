
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)\
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", error,\
				hipGetErrorString(error));\
		exit(EXIT_FAILURE);\
	}\
}

struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start,0);
		hipEventSynchronize(start);
	}

	void Stop()
	{
		hipEventRecord(stop, 0);
	}

	float Elapsed()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};

__global__ void addVecKernel(int *in1, int *in2, int n, 
        int *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    if (i < n)
    {
        out[i] = in1[i] + in2[i];
    }
}

void addVec(int *in1, int *in2, int n, 
        int *out, 
        bool useDevice=false, dim3 blockSize=dim3(1), int nStreams=1)
{
	if (useDevice == false)
	{
        for (int i = 0; i < n; i++)
        {
            out[i] = in1[i] + in2[i];
        }
	}
	else // Use device
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, 0);
		printf("GPU name: %s\n", devProp.name);
		printf("GPU compute capability: %d.%d\n", devProp.major, devProp.minor);
        
        size_t nBytes = n * sizeof(int);

		// TODO: Allocate device memory regions
		int *d_in1, *d_in2, *d_out;
		CHECK(hipMalloc(&d_in1, nBytes));
		CHECK(hipMalloc(&d_in2, nBytes));
		CHECK(hipMalloc(&d_out, nBytes));

        // TODO: Create "nStreams" device streams
		// Create an array of "nStreams" streams on the device
		hipStream_t *streams = (hipStream_t *)malloc(nStreams * sizeof(hipStream_t));
		// Create each stream in the array
		for (int i = 0; i < nStreams; ++i)
		{
			CHECK(hipStreamCreate(&streams[i]));
		}

        GpuTimer timer;
        timer.Start();

        // TODO: Send jobs (H2D, kernel, D2H) to device streams 
		for (int i = 0; i < nStreams; ++i)
		{
			// Number of elements to be processed by the stream
			int streamSize = (n - 1) / nStreams + 1;
			// Calculate the offset of the current stream
			int offset = i * streamSize;

			// Modify the streamSize for the last stream
			if (offset + streamSize > n)
			{
				streamSize = n - offset;
			}

			// Number of bytes to be processed by the stream
			size_t streamBytes = streamSize * sizeof(int);

			// Copy data from host to device for each part asynchronously
			CHECK(hipMemcpyAsync(&d_in1[offset], &in1[offset], streamBytes, hipMemcpyHostToDevice, streams[i]));
			CHECK(hipMemcpyAsync(&d_in2[offset], &in2[offset], streamBytes, hipMemcpyHostToDevice, streams[i]));

			// Calculate the number of blocks needed for each stream
			dim3 gridSize((streamSize - 1) / blockSize.x + 1);

			// Launch the kernel function for each stream asynchronously
			addVecKernel<<<gridSize, blockSize, 0, streams[i]>>>(&d_in1[offset], &d_in2[offset], streamSize, &d_out[offset]);
			CHECK(hipGetLastError()); // Check if there is any error when launching the kernel

			// Copy data from device to host for each part asynchronously
			CHECK(hipMemcpyAsync(&out[offset], &d_out[offset], streamBytes, hipMemcpyDeviceToHost, streams[i]));
		}

        timer.Stop();
        float time = timer.Elapsed();
        printf("Processing time of all device streams: %f ms\n\n", time);

        // TODO: Destroy device streams
		for (int i = 0; i < nStreams; ++i)
		{
			// Make sure that the stream is finished
			CHECK(hipStreamSynchronize(streams[i]));
			// Destroy the stream
			CHECK(hipStreamDestroy(streams[i]));
		}
		// Free the array of streams
		free(streams);

        // TODO: Free device memory regions
		CHECK(hipFree(d_in1));
		CHECK(hipFree(d_in2));
		CHECK(hipFree(d_out));
	}
}

int main(int argc, char ** argv)
{
    int n; 
    int *in1, *in2; 
    int *out, *correctOut;

    // Input data into n
    n = (1 << 24) + 1;
    printf("n =  %d\n\n", n);

    // Allocate memories for in1, in2, out
    size_t nBytes = n * sizeof(int);
    CHECK(hipHostMalloc(&in1, nBytes, hipHostMallocDefault));
    CHECK(hipHostMalloc(&in2, nBytes, hipHostMallocDefault));
    CHECK(hipHostMalloc(&out, nBytes, hipHostMallocDefault));
    correctOut = (int *)malloc(nBytes);

    // Input data into in1, in2
    for (int i = 0; i < n; i++)
    {
    	in1[i] = rand() & 0xff; // Random int in [0, 255]
    	in2[i] = rand() & 0xff; // Random int in [0, 255]
    }

    // Add in1 & in2 on host
    addVec(in1, in2, n, correctOut);

    // Add in1 & in2 on device
	dim3 blockSize(512); // Default
    int nStreams = 1; // Default
	if (argc >= 2)
	{
		blockSize.x = atoi(argv[1]);
        if (argc >= 3)
        {
            nStreams = atoi(argv[2]);
        }
	} 
    addVec(in1, in2, n, out, true, blockSize, nStreams);

    // Check correctness
    for (int i = 0; i < n; i++)
    {
    	if (out[i] != correctOut[i])
    	{
    		printf("INCORRECT :(\n");
    		return 1;
    	}
    }
    printf("CORRECT :)\n");
    
    CHECK(hipHostFree(in1));
    CHECK(hipHostFree(in2));
    CHECK(hipHostFree(out));    
    free(correctOut);
}
