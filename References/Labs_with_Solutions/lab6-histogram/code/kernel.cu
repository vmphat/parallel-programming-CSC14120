
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

// Define your kernels in this file you may use more than one kernel if you
// need to

// INSERT KERNEL(S) HERE

__global__ void histogram_kernel(unsigned int* input, unsigned int* bins,
    unsigned int num_elements, unsigned int num_bins) {

    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;

    // Privatized bins
    extern __shared__ unsigned int bins_s[];
    for(unsigned int binIdx = threadIdx.x; binIdx < num_bins; binIdx +=blockDim.x) {
        bins_s[binIdx] = 0u;
    }
    __syncthreads();

    // Histogram
    for(unsigned int i = tid; i < num_elements; i += blockDim.x*gridDim.x) {
        atomicAdd(&(bins_s[input[i]]), 1);
    }
    __syncthreads();

    // Commit to global memory
    for(unsigned int binIdx = threadIdx.x; binIdx < num_bins; binIdx += blockDim.x) {
        atomicAdd(&(bins[binIdx]), bins_s[binIdx]);
    }

}

__global__ void convert_kernel(unsigned int *bins32, uint8_t *bins8,
    unsigned int num_bins) {

    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;

    if(tid < num_bins) {
        unsigned int count = bins32[tid];
        if (count < 256) {
            bins8[tid] = (uint8_t) count;
        } else {
            bins8[tid] = 255u;
        }
    }


}

/******************************************************************************
Setup and invoke your kernel(s) in this function. You may also allocate more
GPU memory if you need to
*******************************************************************************/
void histogram(unsigned int* input, uint8_t* bins, unsigned int num_elements,
        unsigned int num_bins) {

    // INSERT CODE HERE

    // Create 32 bit bins
    unsigned int *bins32;
    hipMalloc((void**)&bins32, num_bins * sizeof(unsigned int));
    hipMemset(bins32, 0, num_bins * sizeof(unsigned int));

    // Launch histogram kernel using 32-bit bins
    dim3 dim_grid, dim_block;
    dim_block.x = 512; dim_block.y = dim_block.z = 1;
    dim_grid.x = 30; dim_grid.y = dim_grid.z = 1;
    histogram_kernel<<<dim_grid, dim_block, num_bins*sizeof(unsigned int)>>>
        (input, bins32, num_elements, num_bins);

    // Convert 32-bit bins into 8-bit bins
    dim_block.x = 512;
    dim_grid.x = (num_bins - 1)/dim_block.x + 1;
    convert_kernel<<<dim_grid, dim_block>>>(bins32, bins, num_bins);

    // Free allocated device memory
    hipFree(bins32);

}


