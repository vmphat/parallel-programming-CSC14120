#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>

#define FILTER_WIDTH 9
__constant__ float dc_filter[FILTER_WIDTH * FILTER_WIDTH];

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(EXIT_FAILURE);\
    }\
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void readPnm(char * fileName, int &width, int &height, uchar3 * &pixels)
{
	FILE * f = fopen(fileName, "r");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);
	
	if (strcmp(type, "P3") != 0) // In this exercise, we don't touch other types
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	fscanf(f, "%i", &width);
	fscanf(f, "%i", &height);
	
	int max_val;
	fscanf(f, "%i", &max_val);
	if (max_val > 255) // In this exercise, we assume 1 byte per value
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	for (int i = 0; i < width * height; i++)
		fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

	fclose(f);
}

void writePnm(uchar3 * pixels, int width, int height, char * fileName)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	fprintf(f, "P3\n%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height; i++)
		fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);
	
	fclose(f);
}

__global__ void blurImgKernel1(uchar3 * inPixels, int width, int height, 
        float * filter, int filterWidth, 
        uchar3 * outPixels)
{
	// TODO

	/**
	 * Blur the input (RGB) image using GPU
	 * 
	 * Basic kernel function using global memory to access data
	**/

	// Get the position of the current thread in the grid of threads
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;

	// Check if the thread is within the image
	if (r < height && c < width)
	{
		// Calculate the filter radius
		int filterRadius = filterWidth / 2;
		// Initialize the weighted sum for each color channel of the pixel
		float3 sum = make_float3(0.0f, 0.0f, 0.0f);

		// Loop over the filter
		for (int filterR = 0; filterR < filterWidth; ++filterR)
		{
			for (int filterC = 0; filterC < filterWidth; ++filterC)
			{
				// Compute the position of the neighbor pixel in the input image
				int imageR = r + filterR - filterRadius;
				int imageC = c + filterC - filterRadius;
				// Clamp to boundary of the image
				imageR = min(max(imageR, 0), height - 1);
				imageC = min(max(imageC, 0), width  - 1);

				// Get the pixel value at the image position
				uchar3 pixel = inPixels[imageR * width + imageC];
				// Get the filter value at the filter position
				float filterValue = filter[filterR * filterWidth + filterC];
				
				// Accumulate the weighted sum for each color channel
				sum.x += pixel.x * filterValue;
				sum.y += pixel.y * filterValue;
				sum.z += pixel.z * filterValue;
			}
		}

		// Assign the blurred pixel to the output image
		// Clamp the pixel value to [0, 255]
		outPixels[r * width + c] = make_uchar3(
			min(max(int(sum.x), 0), 255),
			min(max(int(sum.y), 0), 255),
			min(max(int(sum.z), 0), 255)
		);
	}
}

__global__ void blurImgKernel2(uchar3 * inPixels, int width, int height, 
        float * filter, int filterWidth, 
        uchar3 * outPixels)
{
	// TODO

	/**
	 * Blur the input (RGB) image using GPU
	 *
	 * This kernel function uses SMEM (shared memory) to reduce the number of
	 * global memory accesses. Each block loads a portion of data from the array 
	 * `inPixels` from GMEM to SMEM. After that, the data stored in SMEM is used 
	 * multiple times for all threads in the block to compute output pixels.
	 *
	 * We will dynamically allocate an array in SMEM for each block. So that, 
	 * array size can change depending on the `filterWidth` and the `blockSize`.
	**/

	// Dynamically allocate an array in SMEM for each block
	extern __shared__ uchar3 s_inPixels[];

	// Get the position of the current thread in the grid of threads
	int outRow = blockIdx.y * blockDim.y + threadIdx.y;
	int outCol = blockIdx.x * blockDim.x + threadIdx.x;
	
	// Calculate the width of the input tile in SMEM
	int filterRadius = filterWidth / 2;
	int inTileWidth  = blockDim.x + 2 * filterRadius;

	// Calculate the position of the current cell in the input tile
	int inTileRow = threadIdx.y + filterRadius;
	int inTileCol = threadIdx.x + filterRadius;

	// Calculate the position of the halo cells in the input tile
	int haloTop    = inTileRow - filterRadius;
	int haloBottom = inTileRow + filterRadius;
	int haloLeft   = inTileCol - filterRadius;
	int haloRight  = inTileCol + filterRadius;

	// Copy data from GMEM to top-left halo cell in SMEM
	if (haloTop < filterRadius || haloLeft < filterRadius)
	{
		// Calculate the position of the pixel in the input image
		// and clamp to boundary of the image
		int inRow = max(outRow - filterRadius, 0);
		int inCol = max(outCol - filterRadius, 0);
		// Copy the input pixel value from GMEM to SMEM
		s_inPixels[haloTop * inTileWidth + haloLeft] = inPixels[inRow * width + inCol];
	}

	// Copy data from GMEM to top-right halo cell in SMEM
	if (haloTop < filterRadius || haloRight >= (filterRadius + blockDim.x))
	{
		// Calculate the position of the pixel in the input image
		// and clamp to boundary of the image
		int inRow = max(outRow - filterRadius, 0);
		int inCol = min(outCol + filterRadius, width - 1);
		// Copy the input pixel value from GMEM to SMEM
		s_inPixels[haloTop * inTileWidth + haloRight] = inPixels[inRow * width + inCol];
	}

	// Copy data from GMEM to bottom-left halo cell in SMEM
	if (haloBottom >= (filterRadius + blockDim.y) || haloLeft < filterRadius)
	{
		// Calculate the position of the pixel in the input image
		// and clamp to boundary of the image
		int inRow = min(outRow + filterRadius, height - 1);
		int inCol = max(outCol - filterRadius, 0);
		// Copy the input pixel value from GMEM to SMEM
		s_inPixels[haloBottom * inTileWidth + haloLeft] = inPixels[inRow * width + inCol];
	}

	// Copy data from GMEM to bottom-right halo cell in SMEM
	if (haloBottom >= (filterRadius + blockDim.y) || haloRight >= (filterRadius + blockDim.x))
	{
		// Calculate the position of the pixel in the input image
		// and clamp to boundary of the image
		int inRow = min(outRow + filterRadius, height - 1);
		int inCol = min(outCol + filterRadius, width  - 1);
		// Copy the input pixel value from GMEM to SMEM
		s_inPixels[haloBottom * inTileWidth + haloRight] = inPixels[inRow * width + inCol];
	}

	// Copy data from GMEM to current cell in SMEM
	if (outRow < height && outCol < width)
	{
		s_inPixels[inTileRow * inTileWidth + inTileCol] = inPixels[outRow * width + outCol];
	}

	// Synchronize to make sure the SMEM is fully populated
	__syncthreads();

	// Check if the thread is within the image
	if (outRow < height && outCol < width)
	{
		// Initialize the weighted sum for each color channel of the pixel
		float3 sum = make_float3(0.0f, 0.0f, 0.0f);

		// Loop over the filter
		for (int filterR = 0; filterR < filterWidth; ++filterR)
		{
			for (int filterC = 0; filterC < filterWidth; ++filterC)
			{
				// Compute the position of the neighbor pixel in the input tile
				int neighborRow = inTileRow + filterR - filterRadius;
				int neighborCol = inTileCol + filterC - filterRadius;

				// Get the pixel value at the input tile position
				uchar3 pixel = s_inPixels[neighborRow * inTileWidth + neighborCol];
				// Get the filter value at the filter position
				float filterValue = filter[filterR * filterWidth + filterC];
				
				// Accumulate the weighted sum for each color channel
				sum.x += pixel.x * filterValue;
				sum.y += pixel.y * filterValue;
				sum.z += pixel.z * filterValue;
			}
		}

		// Assign the blurred pixel to the output image
		// Clamp the pixel value to [0, 255]
		outPixels[outRow * width + outCol] = make_uchar3(
			min(max(int(sum.x), 0), 255),
			min(max(int(sum.y), 0), 255),
			min(max(int(sum.z), 0), 255)
		);
	}
}

__global__ void blurImgKernel3(uchar3 * inPixels, int width, int height, 
        int filterWidth, 
        uchar3 * outPixels)
{
	// TODO

	/**
	 * Blur the input (RGB) image using GPU
	 * 
	 * Like kernel function `blurImgKernel2`, this kernel function will use
	 * SMEM (shared memory) to reduce the number of global memory accesses.
	 * (For more details, please refer to the description of `blurImgKernel2`.)
	 *
	 * However, this kernel function also uses CMEM (constant memory) to store
	 * the filter values. The filter values are read from the global variable
	 * `dc_filter`. By utilizing constant cache, we can reduce the number of
	 * global memory accesses to read the filter values.
	**/

	// Dynamically allocate an array in SMEM for each block
	extern __shared__ uchar3 s_inPixels[];

	// Get the position of the current thread in the grid of threads
	int outRow = blockIdx.y * blockDim.y + threadIdx.y;
	int outCol = blockIdx.x * blockDim.x + threadIdx.x;
	
	// Calculate the width of the input tile in SMEM
	int filterRadius = filterWidth / 2;
	int inTileWidth  = blockDim.x + 2 * filterRadius;

	// Calculate the position of the current cell in the input tile
	int inTileRow = threadIdx.y + filterRadius;
	int inTileCol = threadIdx.x + filterRadius;

	// Calculate the position of the halo cells in the input tile
	int haloTop    = inTileRow - filterRadius;
	int haloBottom = inTileRow + filterRadius;
	int haloLeft   = inTileCol - filterRadius;
	int haloRight  = inTileCol + filterRadius;

	// Copy data from GMEM to top-left halo cell in SMEM
	if (haloTop < filterRadius || haloLeft < filterRadius)
	{
		// Calculate the position of the pixel in the input image
		// and clamp to boundary of the image
		int inRow = max(outRow - filterRadius, 0);
		int inCol = max(outCol - filterRadius, 0);
		// Copy the input pixel value from GMEM to SMEM
		s_inPixels[haloTop * inTileWidth + haloLeft] = inPixels[inRow * width + inCol];
	}

	// Copy data from GMEM to top-right halo cell in SMEM
	if (haloTop < filterRadius || haloRight >= (filterRadius + blockDim.x))
	{
		// Calculate the position of the pixel in the input image
		// and clamp to boundary of the image
		int inRow = max(outRow - filterRadius, 0);
		int inCol = min(outCol + filterRadius, width - 1);
		// Copy the input pixel value from GMEM to SMEM
		s_inPixels[haloTop * inTileWidth + haloRight] = inPixels[inRow * width + inCol];
	}

	// Copy data from GMEM to bottom-left halo cell in SMEM
	if (haloBottom >= (filterRadius + blockDim.y) || haloLeft < filterRadius)
	{
		// Calculate the position of the pixel in the input image
		// and clamp to boundary of the image
		int inRow = min(outRow + filterRadius, height - 1);
		int inCol = max(outCol - filterRadius, 0);
		// Copy the input pixel value from GMEM to SMEM
		s_inPixels[haloBottom * inTileWidth + haloLeft] = inPixels[inRow * width + inCol];
	}

	// Copy data from GMEM to bottom-right halo cell in SMEM
	if (haloBottom >= (filterRadius + blockDim.y) || haloRight >= (filterRadius + blockDim.x))
	{
		// Calculate the position of the pixel in the input image
		// and clamp to boundary of the image
		int inRow = min(outRow + filterRadius, height - 1);
		int inCol = min(outCol + filterRadius, width  - 1);
		// Copy the input pixel value from GMEM to SMEM
		s_inPixels[haloBottom * inTileWidth + haloRight] = inPixels[inRow * width + inCol];
	}

	// Copy data from GMEM to current cell in SMEM
	if (outRow < height && outCol < width)
	{
		s_inPixels[inTileRow * inTileWidth + inTileCol] = inPixels[outRow * width + outCol];
	}

	// Synchronize to make sure the SMEM is fully populated
	__syncthreads();

	// Check if the thread is within the image
	if (outRow < height && outCol < width)
	{
		// Initialize the weighted sum for each color channel of the pixel
		float3 sum = make_float3(0.0f, 0.0f, 0.0f);

		// Loop over the filter
		for (int filterR = 0; filterR < filterWidth; ++filterR)
		{
			for (int filterC = 0; filterC < filterWidth; ++filterC)
			{
				// Compute the position of the neighbor pixel in the input tile
				int neighborRow = inTileRow + filterR - filterRadius;
				int neighborCol = inTileCol + filterC - filterRadius;

				// Get the pixel value at the input tile position
				uchar3 pixel = s_inPixels[neighborRow * inTileWidth + neighborCol];
				// Get the filter value at the filter position
				// ! Read the filter value from CMEM
				float filterValue = dc_filter[filterR * filterWidth + filterC];
				
				// Accumulate the weighted sum for each color channel
				sum.x += pixel.x * filterValue;
				sum.y += pixel.y * filterValue;
				sum.z += pixel.z * filterValue;
			}
		}

		// Assign the blurred pixel to the output image
		// Clamp the pixel value to [0, 255]
		outPixels[outRow * width + outCol] = make_uchar3(
			min(max(int(sum.x), 0), 255),
			min(max(int(sum.y), 0), 255),
			min(max(int(sum.z), 0), 255)
		);
	}
}							

void blurImg(uchar3 * inPixels, int width, int height, float * filter, int filterWidth, 
        uchar3 * outPixels,
        bool useDevice=false, dim3 blockSize=dim3(1, 1), int kernelType=1)
{
	if (useDevice == false)
	{
		//TODO

		/**
		 * Blur the input (RGB) image using CPU
		**/

		// Loop through each pixel in the image
		for (int r = 0; r < height; ++r)
		{
			for (int c = 0; c < width; ++c)
			{
				// Calculate the filter radius
				int filterRadius = filterWidth / 2;
				// Initialize the weighted sum for each color channel of the pixel
				float3 sum = make_float3(0.0f, 0.0f, 0.0f);

				// Loop over the filter
				for (int filterR = 0; filterR < filterWidth; ++filterR)
				{
					for (int filterC = 0; filterC < filterWidth; ++filterC)
					{
						// Compute the position of the neighbor pixel in the input image
						int imageR = r + filterR - filterRadius;
						int imageC = c + filterC - filterRadius;
						// Clamp to boundary of the image
						imageR = min(max(imageR, 0), height - 1);
						imageC = min(max(imageC, 0), width  - 1);

						// Get the pixel value at the image position
						uchar3 pixel = inPixels[imageR * width + imageC];
						// Get the filter value at the filter position
						float filterValue = filter[filterR * filterWidth + filterC];
						
						// Accumulate the weighted sum for each color channel
						sum.x += pixel.x * filterValue;
						sum.y += pixel.y * filterValue;
						sum.z += pixel.z * filterValue;
					}
				}

				// Assign the blurred pixel to the output image
				// Clamp the pixel value to [0, 255]
				outPixels[r * width + c] = make_uchar3(
					min(max(int(sum.x), 0), 255),
					min(max(int(sum.y), 0), 255),
					min(max(int(sum.z), 0), 255)
				);
			}
		}
	}
	else // Use device
	{
		GpuTimer timer;
		
		printf("\nKernel %i, ", kernelType);
		// Allocate device memories
		uchar3 * d_inPixels, * d_outPixels;
		float * d_filter;
		size_t pixelsSize = width * height * sizeof(uchar3);
		size_t filterSize = filterWidth * filterWidth * sizeof(float);
		CHECK(hipMalloc(&d_inPixels, pixelsSize));
		CHECK(hipMalloc(&d_outPixels, pixelsSize));
		if (kernelType == 1 || kernelType == 2)
		{
			CHECK(hipMalloc(&d_filter, filterSize));
		}

		// Copy data to device memories
		CHECK(hipMemcpy(d_inPixels, inPixels, pixelsSize, hipMemcpyHostToDevice));
		if (kernelType == 1 || kernelType == 2)
		{
			CHECK(hipMemcpy(d_filter, filter, filterSize, hipMemcpyHostToDevice));
		}
		else
		{
			// TODO: copy data from "filter" (on host) to "dc_filter" (on CMEM of device)
			CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dc_filter), filter, filterSize));
		}

		// Call kernel
		dim3 gridSize((width-1)/blockSize.x + 1, (height-1)/blockSize.y + 1);
		printf("block size %ix%i, grid size %ix%i\n", blockSize.x, blockSize.y, gridSize.x, gridSize.y);
		timer.Start();
		if (kernelType == 1)
		{
			// TODO: call blurImgKernel1
			blurImgKernel1<<<gridSize, blockSize>>>(d_inPixels, width, height, d_filter, filterWidth, d_outPixels);
		}
		else if (kernelType == 2)
		{
			// TODO: call blurImgKernel2
			size_t smemSize = (blockSize.x + filterWidth) * (blockSize.y + filterWidth) * sizeof(uchar3);
			blurImgKernel2<<<gridSize, blockSize, smemSize>>>(d_inPixels, width, height, d_filter, filterWidth, d_outPixels);
		}
		else
		{
			// TODO: call blurImgKernel3
			size_t smemSize = (blockSize.x + filterWidth) * (blockSize.y + filterWidth) * sizeof(uchar3);
			blurImgKernel3<<<gridSize, blockSize, smemSize>>>(d_inPixels, width, height, filterWidth, d_outPixels);
		}
		timer.Stop();
		float time = timer.Elapsed();
		printf("Kernel time: %f ms\n", time);
		hipDeviceSynchronize();
		CHECK(hipGetLastError());

		// Copy result from device memory
		CHECK(hipMemcpy(outPixels, d_outPixels, pixelsSize, hipMemcpyDeviceToHost));

		// Free device memories
		CHECK(hipFree(d_inPixels));
		CHECK(hipFree(d_outPixels));
		if (kernelType == 1 || kernelType == 2)
		{
			CHECK(hipFree(d_filter));
		}
	}
	
}

float computeError(uchar3 * a1, uchar3 * a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)
	{
		err += abs((int)a1[i].x - (int)a2[i].x);
		err += abs((int)a1[i].y - (int)a2[i].y);
		err += abs((int)a1[i].z - (int)a2[i].z);
	}
	err /= (n * 3);
	return err;
}

void printError(uchar3 * deviceResult, uchar3 * hostResult, int width, int height)
{
	float err = computeError(deviceResult, hostResult, width * height);
	printf("Error: %f\n", err);
}

char * concatStr(const char * s1, const char * s2)
{
    char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

void printDeviceInfo()
{
	hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
    printf("CMEM: %lu bytes\n", devProv.totalConstMem);
    printf("L2 cache: %i bytes\n", devProv.l2CacheSize);
    printf("SMEM / one SM: %lu bytes\n", devProv.sharedMemPerMultiprocessor);
    printf("****************************\n");

}

int main(int argc, char ** argv)
{
	if (argc !=3 && argc != 5)
	{
		printf("The number of arguments is invalid\n");
		return EXIT_FAILURE;
	}

	printDeviceInfo();

	// Read input image file
	int width, height;
	uchar3 * inPixels;
	readPnm(argv[1], width, height, inPixels);
	printf("\nImage size (width x height): %i x %i\n", width, height);

	// Set up a simple filter with blurring effect 
	int filterWidth = FILTER_WIDTH;
	float * filter = (float *)malloc(filterWidth * filterWidth * sizeof(float));
	for (int filterR = 0; filterR < filterWidth; filterR++)
	{
		for (int filterC = 0; filterC < filterWidth; filterC++)
		{
			filter[filterR * filterWidth + filterC] = 1. / (filterWidth * filterWidth);
		}
	}

	// Blur input image not using device
	uchar3 * correctOutPixels = (uchar3 *)malloc(width * height * sizeof(uchar3)); 
	blurImg(inPixels, width, height, filter, filterWidth, correctOutPixels);
	
    // Blur input image using device, kernel 1
    dim3 blockSize(16, 16); // Default
	if (argc == 5)
	{
		blockSize.x = atoi(argv[3]);
		blockSize.y = atoi(argv[4]);
	}	
	uchar3 * outPixels1 = (uchar3 *)malloc(width * height * sizeof(uchar3));
	blurImg(inPixels, width, height, filter, filterWidth, outPixels1, true, blockSize, 1);
	printError(outPixels1, correctOutPixels, width, height);
	
	// Blur input image using device, kernel 2
	uchar3 * outPixels2 = (uchar3 *)malloc(width * height * sizeof(uchar3));
	blurImg(inPixels, width, height, filter, filterWidth, outPixels2, true, blockSize, 2);
	printError(outPixels2, correctOutPixels, width, height);

	// Blur input image using device, kernel 3
	uchar3 * outPixels3 = (uchar3 *)malloc(width * height * sizeof(uchar3));
	blurImg(inPixels, width, height, filter, filterWidth, outPixels3, true, blockSize, 3);
	printError(outPixels3, correctOutPixels, width, height);

    // Write results to files
    char * outFileNameBase = strtok(argv[2], "."); // Get rid of extension
	writePnm(correctOutPixels, width, height, concatStr(outFileNameBase, "_host.pnm"));
	writePnm(outPixels1, width, height, concatStr(outFileNameBase, "_device1.pnm"));
	writePnm(outPixels2, width, height, concatStr(outFileNameBase, "_device2.pnm"));
	writePnm(outPixels3, width, height, concatStr(outFileNameBase, "_device3.pnm"));

	// Free memories
	free(inPixels);
	free(filter);
	free(correctOutPixels);
	free(outPixels1);
	free(outPixels2);
	free(outPixels3);
}
